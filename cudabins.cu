#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include <json/json.h>
#include <assert.h>
#include <jsoncpp.cpp>
#include <iostream>
#include <fstream>
#include <random>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>

using namespace std;
using namespace thrust;

#define BFD
#define SCALE 2

typedef struct obj {
    uint32_t size;
} obj_t;

typedef struct alias {
    int i1;
    int i2;
    float divider;
} alias_t;

typedef struct ghetto_vec {
    __device__
    void push_back(obj_t obj) {
        if (num_entries + 1 >= maxlen) {
            obj_t *old = arr;
            arr = new obj_t[maxlen * SCALE];
            for (int i = 0; i < num_entries; i++) {
                arr[i] = old[i];
            }
            delete[] old;
        }
        arr[num_entries++] = obj;
    }
    __device__
    int size() {
        return num_entries;
    }
    int maxlen;
    int num_entries;
    obj_t *arr;
} ghetto_vec_t;

typedef struct dev_bin {
  uint32_t occupancy;
  ghetto_vec obj_list;
  alias_t alias;
} dev_bin_t;

typedef struct bin {
  uint32_t occupancy;
  host_vector<obj_t> obj_list;
  alias_t alias;
  __host__ ~bin() {}
} bin_t;

typedef struct cudaParams {
    uint32_t total_obj_size;
    uint32_t bin_size;
    uint32_t num_objs;
    obj_t *objs;
} cudaParams;

obj_t *objs;
__constant__ cudaParams params;

obj_t *host_objs;
host_vector<bin_t> host_bins;
uint32_t host_num_objs;
uint32_t host_bin_size;
uint32_t host_total_obj_size = 0; //pseudo-constant
uint32_t host_num_bins;

bin_t *bins_out;

__host__
bool parse(char *infile) {
    ifstream f(infile, std::ifstream::binary);
    if (f.fail()) {
        return false;
    }
    Json::Value obj_data;
    f >> obj_data;
    host_bin_size = obj_data["bin_size"].asUInt();
    host_num_objs = obj_data["num_objs"].asUInt();

    // Initialize object array and put one obj in each bin
    host_objs = new obj_t[host_num_objs];
    auto obj_array = obj_data["objs"];
    for(uint32_t i = 0; i < host_num_objs; i++){
        #ifdef TAGGING
        host_objs[i].tag = i;
        #endif
        host_objs[i].size = obj_array[i].asUInt();
        host_total_obj_size += obj_array[i].asUInt();
    }
    return true;
}

__host__
int calculate_maxsize() {
    const float slip_ratio = .5f;
    int total_size = 0;
    for (size_t i = 0; i < host_num_objs; i++) {
        total_size += host_objs[i].size;
    }
    return (int) ((float) total_size / (slip_ratio * host_bin_size));
}
__device__
void check_bin(dev_bin_t *b, int bin_size) {
    uint32_t sum = 0;
    for (int i = 0; i < b->obj_list.num_entries; i++) {
        sum += b->obj_list.arr[i].size;
    }
    assert(b->occupancy == sum);
    assert(b->occupancy <= bin_size);
}

__global__ void
kernel(dev_bin_t *bins, int maxsize, int *dev_retval_pt) {
    int size = 0;
    int num_objs = params.num_objs;
    obj_t *objs = params.objs;
    int bin_size = params.bin_size;
    // int total_obj_size = params.total_obj_size;
    // printf("bins: %p, maxsize: %i, dev_retval_pt: %p\n",bins,maxsize,dev_retval_pt);
    // printf("num_objs: %i, objs: %p, bin_size: %i, total_obj_size: %i\n",num_objs,objs,bin_size,total_obj_size);
    thrust::sort(cuda::par, objs, &objs[num_objs],
        [](const obj_t &a, const obj_t &b) -> bool { return a.size > b.size; });
    for (size_t i = 0; i < num_objs; i++) {
        obj_t obj = objs[i];
        bool found_fit_flag = false;
        for (size_t j = 0; j < size; j++) {
            dev_bin_t *bin = &bins[j];
            if (bin->occupancy + obj.size <= bin_size) {
                bin->occupancy += obj.size;
                bin->obj_list.push_back(obj);
                found_fit_flag = true;
                break;
            }
            check_bin(bin, bin_size);
        }
        if (!found_fit_flag) {
            dev_bin_t b;
            b.occupancy = obj.size;
            if (size >= maxsize - 1) {
                *dev_retval_pt = -1;
                return;
            }
            b.obj_list.maxlen = 10;
            b.obj_list.arr = new obj_t[b.obj_list.maxlen];
            b.obj_list.num_entries = 0;
            b.obj_list.push_back(obj);
            bins[size] = b;
            size++;
        }
    }
    for (size_t j = 0; j < size; j++) {
        check_bin(&bins[j], bin_size);
    }
    printf("size: %i\n", size);
    *dev_retval_pt = size;
    return;
}

void runBFD(){
    return;
}


#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__host__
void run() {
    dev_bin_t *bins;
    int maxsize = calculate_maxsize();
    int *dev_retval_pt, host_retval;
    cudaParams p;
    gpuErrchk(hipMalloc(&dev_retval_pt, sizeof(int)));
    gpuErrchk(hipMalloc(&bins, maxsize * sizeof(dev_bin_t)));
    gpuErrchk(hipMalloc(&objs, host_num_objs * sizeof(obj_t)));
    gpuErrchk(hipMemcpy(objs, host_objs, host_num_objs * sizeof(obj_t), hipMemcpyHostToDevice));
    p.objs = objs;
    p.num_objs = host_num_objs;
    p.bin_size = host_bin_size;
    p.total_obj_size = host_total_obj_size;
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(params), &p, sizeof(cudaParams)));

    kernel<<<1,1>>>(raw_pointer_cast(&bins[0]), maxsize, dev_retval_pt);
    hipDeviceSynchronize();

    hipMemcpy(&host_retval, dev_retval_pt, sizeof(int), hipMemcpyDeviceToHost);
    if (host_retval < 0) {
        cout << "CUDA kernel failed to pack bins\n";
    }
    host_num_bins = host_retval;
    bins_out = new bin_t[host_num_bins];
    for (size_t i = 0; i < host_num_bins; i++) {
        int objs_in_bin;
        hipMemcpy(&objs_in_bin, &bins[i].obj_list.num_entries,
                   sizeof(int), hipMemcpyDeviceToHost);
        bin_t *b = new bin_t;
        b->obj_list.resize(objs_in_bin);

        hipMemcpy(&b->obj_list[0], &bins[i].obj_list.arr,
                   objs_in_bin * sizeof(obj_t), hipMemcpyDeviceToHost);
        hipMemcpy(&b->occupancy, &bins[i].occupancy,
                   sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(&b->alias, &bins[i].alias,
                   sizeof(alias_t), hipMemcpyDeviceToHost);
        bins_out[i] = *b;
    }
}
__host__
bool dump(char *outfile) {
    Json::Value obj_data;
    obj_data["bin_size"] = host_bin_size;
    obj_data["num_objs"] = host_num_objs;
    obj_data["num_bins"] = host_num_bins;
    obj_data["objs"] = Json::Value(Json::arrayValue);
    obj_data["bins"] = Json::Value(Json::arrayValue);
    for(uint32_t i = 0; i < host_num_objs; i++){
        obj_data["objs"][i] = host_objs[i].size;
    }
    for(uint32_t i = 0; i < (uint32_t) host_num_bins; i++) {
        bin_t bin = bins_out[i];
        obj_data["bins"][i] = Json::Value(Json::arrayValue);
        for (uint32_t j = 0; j < bin.obj_list.size(); j++) {
            obj_data["bins"][i][j] = bin.obj_list[j].size;
        }
    }
    if (outfile==NULL) { //print results to stdout
        cout << "num_objs: " << host_num_objs << endl;
        cout << "num_bins: " << host_num_bins << endl;
    } else { //print to file
        filebuf fb;
        fb.open(outfile, ios::out);
        ostream f(&fb);
        f << obj_data;
    }
    delete[] host_objs;
    return true;
}
