#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include <json/json.h>
#include <assert.h>
#include <jsoncpp.cpp>
#include <iostream>
#include <fstream>
#include <random>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>

#include "cudabins.h"

using namespace std;
using namespace thrust;


typedef struct obj {
    uint32_t size;
}obj;

struct alias {
    int i1;
    int i2;
    float divider;
};


struct dev_bin {
  uint32_t occupancy;
  ghetto_vec<obj> obj_list;
  alias alias;
};

struct bin {
  uint32_t occupancy;
  host_vector<obj> obj_list;
  alias alias;
  __host__ ~bin() {}
};

struct cudaParams {
    uint32_t total_obj_size;
    uint32_t bin_size;
    uint32_t num_objs;
    obj *objs;
};

obj *objs;
__constant__ cudaParams params;

obj *host_objs;
host_vector<bin> host_bins;


uint32_t host_num_objs;
uint32_t host_bin_size;
uint32_t host_total_obj_size = 0; //pseudo-constant
uint32_t host_num_bins;

bin *bins_out;

__host__
bool parse(char *infile) {
    ifstream f(infile, std::ifstream::binary);
    if (f.fail()) {
        return false;
    }
    Json::Value obj_data;
    f >> obj_data;
    host_bin_size = obj_data["bin_size"].asUInt();
    host_num_objs = obj_data["num_objs"].asUInt();

    // Initialize object array and put one obj in each bin
    host_objs = new obj[host_num_objs];
    auto obj_array = obj_data["objs"];
    for(uint32_t i = 0; i < host_num_objs; i++){
        #ifdef TAGGING
        host_objs[i].tag = i;
        #endif
        host_objs[i].size = obj_array[i].asUInt();
        host_total_obj_size += obj_array[i].asUInt();
    }
    return true;
}

__host__
int calculate_maxsize() {
    const float slip_ratio = .5f;
    int total_size = 0;
    for (size_t i = 0; i < host_num_objs; i++) {
        total_size += host_objs[i].size;
    }
    return (int) ((float) total_size / (slip_ratio * host_bin_size));
}

__device__
void check_bin(dev_bin *b, int bin_size) {
    uint32_t sum = 0;
    for (int i = 0; i < b->obj_list.size(); i++) {
        sum += b->obj_list.arr[i].size;
    }
    assert(b->occupancy == sum);
    assert(b->occupancy <= bin_size);
}

void host_check_bin(bin *b, size_t bin_size) {
    uint32_t sum = 0;
    for (size_t i = 0; i < b->obj_list.size(); i++) {
        sum += b->obj_list[i].size;
    }
    assert(b->occupancy == sum);
    assert(b->occupancy <= bin_size);
}


// Recalculate data structures used by rand_empty & _full based on current bins.
__device__
void setup_rand(int size, dev_bin *bins,
                ghetto_vec<float> ecdfs, ghetto_vec<float> fcdfs){
    ecdfs.resize(size);
    fcdfs.resize(size);
    int bin_size = params.bin_size;
    int total_obj_size = params.total_obj_size;

    float sum_empty_space = (float)(size * bin_size - total_obj_size);
    float ecdf = 0.f;
    float fcdf = 0.f;
    for(uint32_t i = 0; i < size; i++){
        ecdf += ((float) (bin_size - bins[i].occupancy)) / sum_empty_space;
        ecdfs[i] = ecdf;
        fcdf += ((float) bins[i].occupancy) / total_obj_size;
        fcdfs[i] = fcdf;
    }
}


__global__ void
kernel(dev_bin *bins, int maxsize, int *dev_retval_pt,
       obj *obj_out, size_t *idx_out) {
    size_t size = 0;
    int num_objs = params.num_objs;
    obj *objs = params.objs;
    int bin_size = params.bin_size;
    ghetto_vec<float> ecdfs; // CDF of empty space
    ghetto_vec<float> fcdfs; // CDF of full space
    thrust::sort(cuda::par, objs, &objs[num_objs],
        [](const obj &a, const obj &b) -> bool { return a.size > b.size; });
    for (size_t i = 0; i < num_objs; i++) {
        obj obj = objs[i];
        bool found_fit_flag = false;
        for (size_t j = 0; j < size; j++) {
            dev_bin *bin = &bins[j];
            if (bin->occupancy + obj.size <= bin_size) {
                bin->occupancy += obj.size;
                bin->obj_list.push_back(obj);
                found_fit_flag = true;
                break;
            }
            check_bin(bin, bin_size);
        }
        if (!found_fit_flag) {
            dev_bin b;
            b.occupancy = obj.size;
            if (size >= maxsize - 1) {
                *dev_retval_pt = -1;
                return;
            }
            b.obj_list.push_back(obj);
            bins[size] = b;
            size++;
        }
    }

    // Copy objects to serial output
    size_t out_idx = 0;
    size_t bi;
    for(bi = 0; bi < size; bi++){
      idx_out[bi] = out_idx;
      for(size_t oi = 0; oi < bins[bi].obj_list.size(); oi++){
        obj_out[out_idx] = bins[bi].obj_list.arr[oi];
        out_idx++;
      }
    }
    idx_out[bi] = out_idx;

    for (size_t j = 0; j < size; j++) {
        check_bin(&bins[j], bin_size);
    }
    printf("Num bins: %zu\n", size);
    *dev_retval_pt = (int) size;
    return;
}

void runBFD(){
    return;
}


#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__host__
void run() {
    dev_bin *bins;
    obj *obj_out;
    size_t *idx_out;
    int maxsize = calculate_maxsize();
    cout << "Max number of bins " << maxsize << std::endl;
    int *dev_retval_pt, host_retval;
    cudaParams p;
    gpuErrchk(hipMalloc(&dev_retval_pt, sizeof(int)));
    gpuErrchk(hipMalloc(&bins, maxsize * sizeof(dev_bin)));
    gpuErrchk(hipMalloc(&objs, host_num_objs * sizeof(obj)));
    gpuErrchk(hipMalloc(&obj_out, host_num_objs * sizeof(obj)));
    gpuErrchk(hipMalloc(&idx_out, host_num_objs * sizeof(size_t)));
    gpuErrchk(hipMemcpy(objs, host_objs, host_num_objs * sizeof(obj), hipMemcpyHostToDevice));
    p.objs = objs;
    p.num_objs = host_num_objs;
    p.bin_size = host_bin_size;
    p.total_obj_size = host_total_obj_size;
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(params), &p, sizeof(cudaParams)));

    kernel<<<1,1>>>(raw_pointer_cast(&bins[0]), maxsize, dev_retval_pt,
                    obj_out, idx_out);
    hipDeviceSynchronize();

    hipMemcpy(&host_retval, dev_retval_pt, sizeof(int), hipMemcpyDeviceToHost);
    if (host_retval < 0) {
        cout << "CUDA kernel failed to pack bins\n";
    }
    host_num_bins = host_retval;
    bins_out = new bin[host_num_bins];

    // Copy the representation of objs in bins to host
    size_t *host_idxs = new size_t[host_num_bins+1];
    obj *host_objs = new obj[host_num_objs];
    hipMemcpy(host_idxs, idx_out, (host_num_bins + 1) * sizeof(size_t),
               hipMemcpyDeviceToHost);
    hipMemcpy(host_objs, obj_out, (host_num_objs) * sizeof(obj),
               hipMemcpyDeviceToHost);

    for (size_t bi = 0; bi < host_num_bins; bi++) {
        bin *b = new bin;
        for(size_t oi = host_idxs[bi]; oi < host_idxs[bi + 1]; oi++){
          b->obj_list.push_back(host_objs[oi]);
        }
        bins_out[bi] = *b;
    }
}

__host__
bool dump(char *outfile) {
    Json::Value obj_data;
    obj_data["bin_size"] = host_bin_size;
    obj_data["num_objs"] = host_num_objs;
    obj_data["num_bins"] = host_num_bins;
    obj_data["objs"] = Json::Value(Json::arrayValue);
    obj_data["bins"] = Json::Value(Json::arrayValue);
    for(uint32_t i = 0; i < host_num_objs; i++){
        obj_data["objs"][i] = host_objs[i].size;
    }
    for(uint32_t i = 0; i < (uint32_t) host_num_bins; i++) {
        bin bin = bins_out[i];
        obj_data["bins"][i] = Json::Value(Json::arrayValue);
        for (uint32_t j = 0; j < bin.obj_list.size(); j++) {
            obj_data["bins"][i][j] = bin.obj_list[j].size;
        }
    }
    if (outfile==NULL) { //print results to stdout
        cout << "num_objs: " << host_num_objs << endl;
        cout << "num_bins: " << host_num_bins << endl;
    } else { //print to file
        filebuf fb;
        fb.open(outfile, ios::out);
        ostream f(&fb);
        f << obj_data;
    }
    delete[] host_objs;
    return true;
}
